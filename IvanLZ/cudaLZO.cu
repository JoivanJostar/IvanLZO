#include "hip/hip_runtime.h"
#include "cuda_lzo.h"
#include "global_var.h"
#include <Windows.h>
#include <iostream>
#include "Console.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h "




__device__ uint32_t  get4Byte(volatile const unsigned char* src) {
    uint32_t result = 0;
    if (((uint64_t)src & (uint64_t)(3)) == 0) {
        // printf("p %p 4�ֽڶ���\n", src);
        return *(volatile int*)src;
    }
    else if (((uint64_t)src & (uint64_t)(1)) == 0) {
        ((unsigned short*)&result)[0] = ((unsigned short*)src)[0];
        ((unsigned short*)&result)[1] = ((unsigned short*)src)[1];
        return result;
    }
    else {//�����Ż�����2�ֽڶ���ʱ�ļ���
     //   printf("p %p ����4�ֽڶ���\n", src);
        ((unsigned char*)&result)[0] = src[0];
        ((unsigned char*)&result)[1] = src[1];
        ((unsigned char*)&result)[2] = src[2];
        ((unsigned char*)&result)[3] = src[3];
        //printf("���ֽڷ���4�ֽڵ�Ԫ�ɹ�,4�ֽ�����Ϊ %u\n", result);
        return result;
    }
}
__device__ lzo_uint device_do_compress(const lzo_bytep in, lzo_uint  in_len,
    lzo_bytep out, lzo_uintp out_len,
    lzo_uint  ti, lzo_voidp wrkmem)
{
    const lzo_bytep ip;
    lzo_bytep op;
    const lzo_bytep const in_end = in + in_len;
    const lzo_bytep const ip_end = in + in_len - 20; //ll>20��-20�ǰ�ȫ��
    const lzo_bytep ii;
    lzo_dict_p const dict = (lzo_dict_p)wrkmem;

    op = out;
    ip = in;
    ii = ip;

    ip += ti < 4 ? 4 - ti : 0;
    for (;;)
    {

        const lzo_bytep m_pos;
        //��Deterministicģʽ�� 2�ι�ϣ���ԣ�����Ĭ�ϲ�����


        lzo_uint m_off;
        lzo_uint m_len;
        {
            lzo_uint32_t dv;
            lzo_uint dindex;
            literal:
            ip += 1 + ((ip - ii) >> 5); //����ط���ȡ���˴��Ż����Ӵ�ѹ���̶�
            //ip += 1;

        next:

            if (ip >= ip_end) //���������˷�Χ����Ҫ����
                break;
            // printf("try GET_LE32 from %p \n", ip);
            dv = get4Byte(ip);
            //dv = UA_GET_LE32(ip); //ȡip��4�ֽڵ����ݵ�dv
           // printf("GET_LE32 sucess dv is %u\n",dv);
            dindex = DINDEX(dv, ip);//get hashKey
            m_pos = in + dict[dindex];


            dict[dindex] = (unsigned short int) ((lzo_uint)(ip - in));

            if (dv != get4Byte(m_pos))//��ײ ����ֱ�ӵ�������������
                goto literal;
        }

        //�������ַ� ��������ַ�������+���ַ�1�����ַ�2�����ַ�3
        ii -= ti; ti = 0; //����iiָ�룬����һ�����ݿ�ĩβδ������ַ���
        {
            lzo_uint t = ip - ii;

            if (t != 0)
            {
                if (t <= 3)
                {
                    op[-2] = LZO_BYTE(op[-2] | t);

                    { do *op++ = *ii++; while (--t > 0); } //�����������֧�ַǶ������ݷ��� ��ʱ��Ҫ����ʹ�����ֽڷ���ָ�Ч�ʽ������
                }

                else
                {
                    if (t <= 18)
                        *op++ = LZO_BYTE(t - 3); //��ʾ���ַ��������Ǹ��ֽڵ�ֵҪС��16���ͱ�ʾ�ظ��ֶε��ֽ����ֿ�������ΪС��3ʱ��piggyback,
                    //���Դ���3ʱ �������޿��Զ����ʾ3���ַ���������18���ַ�
                    else
                    {
                        lzo_uint tt = t - 18; //����ʱҪ+18
                        *op++ = 0;
                        while (tt > 255)
                        {
                            tt -= 255;
                            UA_SET1(op, 0);//Ӧ��û���⣿1�ֽ��Ƕ����
                            op++;
                        }

                        *op++ = LZO_BYTE(tt);
                    }

                    {
                        do *op++ = *ii++; while (--t > 0);
                    }
                }
            }
        }
        m_len = 4; //���������������ڼ����ظ�����len
        {
            //���صĳ���ƥ���㷨�������ϵ����ô������ΪGPU���ܲ�֧�ַǶ�����ʣ����ﵥ�ֽڷô����ȫ�ġ�
            // CPU�Ͽ�����UA���������١�
                      // printf("���Գ���ƥ��\n");
            if (ip[m_len] == m_pos[m_len]) {
                do {
                    m_len += 1;
                    if (ip[m_len] != m_pos[m_len])
                        break;
                    m_len += 1;
                    if (ip[m_len] != m_pos[m_len])
                        break;
                    m_len += 1;
                    if (ip[m_len] != m_pos[m_len])
                        break;
                    m_len += 1;
                    if (ip[m_len] != m_pos[m_len])
                        break;
                    m_len += 1;
                    if (ip[m_len] != m_pos[m_len])
                        break;
                    m_len += 1;
                    if (ip[m_len] != m_pos[m_len])
                        break;
                    m_len += 1;
                    if (ip[m_len] != m_pos[m_len])
                        break;
                    m_len += 1;
                    if (ip + m_len >= ip_end)
                        goto m_len_done;
                } while (ip[m_len] == m_pos[m_len]);
            }
            // printf("�ظ�����ƥ��ɹ�\n");
        }
    m_len_done:
        m_off = ip - m_pos;//get Distance
        ip += m_len;//����ip
        ii = ip;//����ii
        //M2:len<=8 && distance<=2k   
        if (m_len <= M2_MAX_LEN && m_off <= M2_MAX_OFFSET)
        {
            m_off -= 1; //m_off ��m_lenʵ�ʴ洢��Χ[1,8] ��[1,2048]
            *op++ = LZO_BYTE(((m_len - 1) << 5) | ((m_off & 7) << 2));
            *op++ = LZO_BYTE(m_off >> 3);
        }
        //M3:distance<=16k   
        else if (m_off <= M3_MAX_OFFSET)
        {
            m_off -= 1;
            if (m_len <= M3_MAX_LEN)
                *op++ = LZO_BYTE(M3_MARKER | (m_len - 2)); //M3��һ���ֽڵĵ�5λΪlen��len����ʾ��ΧΪ31������Ϊlen��Ȼ����2���������ޱ�ʾ��Χ+2
            //���ǵ�m_len����>=4,�����ΧΪ[4,33]
            else
            {
                m_len -= M3_MAX_LEN; //����ʱ+33
                *op++ = M3_MARKER | 0;
                while (m_len > 255)
                {
                    m_len -= 255;
                    UA_SET1(op, 0);
                    op++;
                }//end of loop
                *op++ = LZO_BYTE(m_len);
            }
            *op++ = LZO_BYTE(m_off << 2);
            *op++ = LZO_BYTE(m_off >> 6);
        }
        //M4:16K<distance<=48k
        else
        {
            m_off -= 0x4000; //��Ϊ֪���������Ҳ��16k���ʿ��Զ���ȥ16k�����ʾ��Χ
            if (m_len <= M4_MAX_LEN) //������3bit��ʾ
                *op++ = LZO_BYTE(M4_MARKER | ((m_off >> 11) & 8) | (m_len - 2));
            else
            {
                m_len -= M4_MAX_LEN;//����ʱ��ԭ
                *op++ = LZO_BYTE(M4_MARKER | ((m_off >> 11) & 8));
                while (m_len > 255)
                {
                    m_len -= 255;
                    UA_SET1(op, 0);
                    op++;
                }
                *op++ = LZO_BYTE(m_len);
            }
            *op++ = LZO_BYTE(m_off << 2);
            *op++ = LZO_BYTE(m_off >> 6);
        }
        goto next;
    }

    *out_len = op - out;
    return in_end - (ii - ti);
}

__device__ int CUDA_COMPRESS_LZO(const lzo_bytep in, lzo_uint  in_len,
    lzo_bytep out, lzo_uintp out_len,
    unsigned char* wrkmem)
{
    const lzo_bytep ip = in;
    lzo_bytep op = out;
    lzo_uint l = in_len;
    lzo_uint t = 0;
    while (l > 20)
    {
        lzo_uint ll = l;
        lzo_uint ll_end;
#if 0 || (LZO_DETERMINISTIC)
        ll = LZO_MIN(ll, 49152); //DETERMINISTICģʽ�� src��������С>48kʱ�ֿ鴦��
#endif
        ll_end = (lzo_uint)ip + ll;
        if ((ll_end + ((t + ll) >> 5)) <= ll_end || (const lzo_bytep)(ll_end + ((t + ll) >> 5)) <= ip + ll)
            break;
#if (LZO_DETERMINISTIC)

        //�޸Ĺ���memset�߼�
        //int size = (lzo_uint)1 << D_BITS;
        //Reset��ϣ�� ���룡������
        unsigned short int* p = (unsigned short int*)wrkmem;
        for (int i = 0; i <1u<<D_BITS; ++i) {
            p[i] = 0;
        }

        //memset(wrkmem, 0, ((lzo_uint)1 << D_BITS) * sizeof(lzo_dict_t)); //
#endif
        //printf("���� do_compress\n");
        t = device_do_compress(ip, ll, op, out_len, t, wrkmem);
        __syncthreads();
        // printf("do_compress�ɹ�\n");
        ip += ll;
        op += *out_len;
        l -= ll;
    }
    t += l;
    // printf("���Դ���߽���\n");
    if (t > 0)
    {
        __syncthreads();
        const lzo_bytep ii = in + in_len - t;

        if (op == out && t <= 238) //sp��֧
            *op++ = LZO_BYTE(17 + t);//���ҽ���do_compressû������κ�����ѹ����opָ��û���ƶ����ṩ�����ݿ�û���κο����滻���ظ��ֶΣ��Ż����˷�֧
        else if (t <= 3)
            op[-2] = LZO_BYTE(op[-2] | t);
        else if (t <= 18)
            *op++ = LZO_BYTE(t - 3);
        else
        {
            lzo_uint tt = t - 18;

            *op++ = 0;
            while (tt > 255)
            {
                tt -= 255;
                *op++ = 0;
                //op++;
            }

            *op++ = LZO_BYTE(tt);
        }
        do
        {
            *op++ = *ii++;
        } while (--t > 0);

       // op += t;ע�͹�
    }
    // printf("�߽��ϴ������\n");
    *op++ = M4_MARKER | 1; //17
    *op++ = 0;
    *op++ = 0;

    *out_len = op - out;
    // printf("LZOѹ��ִ����� outlen=%ld\n",*out_len);
    return LZO_E_OK;
}


__device__ int CUDA_DECOMPRESS_LZO(const lzo_bytep in, lzo_uint  in_len,
    lzo_bytep out, lzo_uintp out_len)
{

    lzo_bytep op;
    const lzo_bytep ip;
    lzo_uint t;
    const lzo_bytep m_pos;
    const lzo_bytep const ip_end = in + in_len;
    *out_len = 0;

    op = out;
    ip = in;

    if (*ip > 17) //���������������ѹ�����ݳ���t<238��ȫ�ǲ����滻���ݣ���ʱѹ��������ݵ�һ���ֽ�=t+17,��ʾ֮����t�����ַ�
    {
        t = *ip++ - 17;//-17����
        if (t < 4)
            goto match_next;
        //if t>4
        do *op++ = *ip++; while (--t > 0);
        goto first_literal_run;
    }

    for (;;)
    {
        t = *ip++;
        if (t >= 16)
            goto match; //���ظ��ֶΣ���ת��match����
        if (t == 0)//�����ַ��ֶΣ����Ҹ���>18 t��Ҫ+18����
        {
            while (*ip == 0)
            {
                t += 255;
                ip++;
            }
            t += 15 + *ip++; //t����3
        }

        //�����ַ��ֶΣ����Ǹ���<18, ��Ҫ+3���� NEED t+3
        {
            *op++ = *ip++; *op++ = *ip++; *op++ = *ip++; //����t+3
            do *op++ = *ip++; while (--t > 0);
        }
        //��һ�� literal �滻
    first_literal_run:
        t = *ip++; //ȡ�ظ��ֶε�һ���ֽ�
        if (t >= 16) //��������ظ��ֶεĵ�һ���ֽھ��Ǵ��ڵ���16
            goto match;//��ת��ƥ���滻����



        //BEG�����������������֧��Ӧ����LZOX����
        printf("�����˴���ķ�֧,�ļ�������\n");
        m_pos = op - (1 + M2_MAX_OFFSET);
        m_pos -= t >> 2;
        m_pos -= *ip++ << 2;
        *op++ = *m_pos++; *op++ = *m_pos++; *op++ = *m_pos;
        goto match_done;
        //END�����������������֧��Ӧ����LZOX����


        for (;;)
        {
        match:
            //M2
            if (t >= 64)
            {

                m_pos = op - 1;//op-1 ��Ϊdistance�洢ʱ-1�ˣ���������op-1��Ϊ�˻�ԭdistance+1����Ϊ����������-1��
                m_pos -= (t >> 2) & 7;
                m_pos -= *ip++ << 3;

                t = (t >> 5) - 1;//need_ip(2)

                goto copy_match;
            }
            //M3
            else if (t >= 32)
            {
                t &= 31;//t-=33; need(2)
                if (t == 0)
                {
                    while (*ip == 0)
                    {
                        t += 255;
                        ip++;
                    }
                    t += 31 + *ip++;
                }
                m_pos = op - 1;
                m_pos -= (ip[0] >> 2) + (ip[1] << 6);
                ip += 2;
            }
            //M4
            else if (t >= 16)
            {
                m_pos = op;//ѹ����ʱ��m_offû��-1,�������ﲻ��m_pos=op-1
                m_pos -= (t & 8) << 11;
                t &= 7;
                if (t == 0)
                {
                    while (*ip == 0)
                    {
                        t += 255;
                        ip++;

                    }
                    t += 7 + *ip++;//t����2��
                }
                m_pos -= (ip[0] >> 2) + (ip[1] << 6);
                ip += 2;
                if (m_pos == op)//distance=0; ������־
                    goto eof_found;
                m_pos -= 0x4000;
            }
            //δ����ķ�֧�������������в�Ӧ�ý���˷�֧�� ��Ϊt>=16�Ǳ�Ȼ��
            else
            {
                m_pos = op - 1;
                m_pos = op - 1;
                m_pos -= t >> 2;
                m_pos -= *ip++ << 2;
                *op++ = *m_pos++; *op++ = *m_pos;
                goto match_done;
            }

            //main CopyMatch:

            {
            copy_match:
                *op++ = *m_pos++; *op++ = *m_pos++;//����m_len+2
                do *op++ = *m_pos++; while (--t > 0);
            }


        match_done: //�ظ��ֶδ�����ϣ�����������Ƿ����piggyback������������match_next�������˳�matchѭ���壬������ѭ����ִ�з�piggyback���ַ��ֶΡ�
            t = ip[-2] & 3;
            if (t == 0)
                break;
            //��piggyback 
        match_next:
            // assert(t > 0); assert(t < 4);
#if 0
            do *op++ = *ip++; while (--t > 0);
#else
            * op++ = *ip++;
            if (t > 1) { *op++ = *ip++; if (t > 2) { *op++ = *ip++; } }
#endif
            t = *ip++;//��һ�ظ��ֶΣ�����matchѭ��
        }
    }

eof_found:
    *out_len = op - out;
    return (ip == ip_end ? LZO_E_OK : LZO_E_ERROR);

}


__global__ void cudaLZ(EncodeResource r) {
 
    //__shared__ unsigned char hash[32 * 1024];
    int tid = blockDim.x * blockIdx.x + threadIdx.x;//��ȡ��ǰ�߳�id
    unsigned char* mybegin = r.in + tid * r.block_size;
    size_t  myinlen = 0;
    if (r.inlen < tid * r.block_size)
        return;

    if (tid == r.remainTid) {
        if (r.remainSize != 0) {
            //printf("cuda�߳�%d Ϊ�߽����߳� �����������Ϊ %d\n", tid, r.remainSize);
            myinlen = r.remainSize;
        }
        else {
            myinlen = r.block_size;//����Ҫ�߽����߳�
        }

    }
    else {
        myinlen = r.block_size;
    }
    __syncthreads();
    //    hipDeviceSynchronize();
    unsigned char* myout = r.out + tid * r.max_out_len;
    unsigned char* myhashtable = r.hashtable + tid * LZO_WRKMEM_SIZE;
    //myhashtable = hash + threadIdx.x * LZO_WRKMEM_SIZE;
    size_t* myoutlen = r.outlenArray + tid;
    *myoutlen = 0;
    CUDA_COMPRESS_LZO(mybegin, myinlen, myout, myoutlen, myhashtable);
   // printf("1\n");

    return;
}



void PrintGPUInfo(hipDeviceProp_t& Prop) {
    //cout<<"GPU:"
    printf("%25s: %s\n", "����",Prop.name);
    printf("%25s: %d\n", "SM������������", Prop.multiProcessorCount);
    printf("%25s: %d KHZ\n", "����Ƶ��", Prop.clockRate);
    printf("%25s: %d\n", "SM�߳�����С(", Prop.warpSize);
    printf("%25s: %d /Per Block\n", "����߳�����", Prop.maxThreadsPerBlock);
    printf("%25s: %d /Per Grid\n", "��������� (��1ά)", Prop.maxGridSize[0]);
    printf("%25s: %d /Per Grid\n", "��������� (��2ά)", Prop.maxGridSize[1]);
    printf("%25s: %d /Per Grid\n", "��������� (��3ά)", Prop.maxGridSize[2]);
    printf("%25s: %d MBytes\n", "GDDR5ȫ���ڴ��С", Prop.totalGlobalMem/1024/1024);
    printf("%25s: %d KBytes/Per Block\n", "������������ڴ��С", Prop.sharedMemPerBlock/1024);
    printf("%25s: %d KHZ\n", "�ڴ�Ƶ��", Prop.memoryClockRate);
    printf("%25s: %d KBytes\n", "SM����L2�����С", Prop.l2CacheSize/1024);
    printf("\n");

}

size_t getFileLength(string filePath) {
    ifstream infile(filePath, ifstream::binary);
    if (!infile) {
        infile.close();
        cout << "I/O Exception  No such File :" + filePath << endl;
        return -1;
    }
    infile.seekg(0, infile.end);
    size_t file_length = infile.tellg();
    infile.seekg(0, infile.beg);
    return file_length;
}

void InitGPU() {
    int NumsGpus = 0;
    hipError_t cudaStatus;
    hipDeviceProp_t Prop;
    Console::ClearScreen();
    cout << "���ڲ�ѯ�Կ���Ϣ������������\n";
    Sleep(500);
    checkCudaErrors(hipGetDeviceCount(&NumsGpus));
    if (NumsGpus == 0) {
        cout << "\n No NVIDA CUDA Device is available\n";
        cout << "��ǰ�豸��֧��NVIDA CUDA �޿���NVIDA�Կ�\n";
        exit(-1);
    }
    for (int i = 0; i < NumsGpus; ++i) {
        cudaStatus = hipGetDeviceProperties(&Prop, i);
        if (cudaStatus != hipSuccess) {
            cout << "Fail to get cuda device properties\n";
            exit(0);
        }
        printf("GPU \[%d\]:\n\n", i);
        PrintGPUInfo(Prop);
    }
    string select;
    int gpu_index;
select_gpu:
    cout << "\nѡ����Ҫ���е�GPU\n";
    cin >> select;
    try {
        gpu_index = stoul(select.c_str());
    }
    catch (std::invalid_argument e) {
        cout << "���������\n";
        goto select_gpu;
    }
    if (!(0 <= gpu_index && gpu_index < NumsGpus)) {
        cout << "���������\n";
        goto select_gpu;
    }


    checkCudaErrors(hipGetDeviceProperties(&Prop, gpu_index));//��ȡ0��GPU����Ϣ

    checkCudaErrors(hipSetDevice(gpu_index)); //ѡ����select��GPU��ִ�б�����

}

  
//��������ִ�еĺ�����
int CallCudaLZO_Encode(unsigned char* in_CPU, size_t inlen, vector<outputUint>& out_CPU, int N, size_t blocksize) {
    unsigned char* GPU_in = NULL;
    unsigned char* GPU_out = NULL;
    unsigned short int* GPU_HashTable = NULL;
    size_t* GPU_outlenArray = NULL;
    hipError_t error;
    float elapsedTime = 0.0f;
    int remainSize = inlen % blocksize;//ʣ��߽��ϴ�С
    int threadNum = (inlen + blocksize - 1) / (blocksize);//ȡ���� 
    int remainTid = threadNum - 1;//����߽����̵߳�id����remainSize=0ʱ����Ҫ�߽����߳�

    //���߳���������һ��
    int blockdim = g_config.block_dim;
    threadNum = ALIGNE_N(threadNum, blockdim);
    EncodeResource r = { 0 };

    
        error = hipMalloc(&GPU_in, inlen + 1024);

        if (error != hipSuccess) {
            printf("WARNING::�ڴ����ʧ�ܣ��Դ治��,�볢��ʹ��cpuѹ��\n");
            return -1;
        }
        //[512k][512k].....[512k][remain]
        int max_out_len_per_thread = ( 48*1024 + LZO_SAFE_OUT_LEN(blocksize));
        int GPU_out_buffer_size = max_out_len_per_thread * threadNum;
        error = hipMalloc(&GPU_out, GPU_out_buffer_size);
        //[LZO_SAFE(512k)][pading][LZO_SAFE(512k)][pading]......[LZO_SAFE(remain)][pading]
        if (error != hipSuccess) {
            printf("WARNING::�ڴ����ʧ�ܣ��Դ治��,�볢��ʹ��cpuѹ��\n");
            hipFree(GPU_in);
            return -1;
        }
        finish_rate = 0.2f;
        error = hipMalloc(&GPU_HashTable, LZO_WRKMEM_SIZE * threadNum);//ÿ���߳�һ��������16k*2=32k�ֽڹ�ϣ��
        //[hash32k0][hash32k].....[hash32k N-1]
        if (error != hipSuccess) {
            printf("WARNING::�ڴ����ʧ�ܣ��Դ治��,�볢��ʹ��cpuѹ��\n");
            hipFree(GPU_in);
            hipFree(GPU_out);
            return -1;
        }
        error = hipMalloc(&GPU_outlenArray, sizeof(size_t) * threadNum);//ÿ���߳�һ��outlen
        //outlen0,outlen1,outlen2,.....outlen N-1
        if (error != hipSuccess) {
            printf("WARNING::�ڴ����ʧ�ܣ��Դ治��,�볢��ʹ��cpuѹ��\n");
            hipFree(GPU_in);
            hipFree(GPU_out);
            hipFree(GPU_HashTable);
            return -1;
        }

        //������ʼ��
        //�������ѹ�����ݴ��ݵ��Դ�
        checkCudaErrors(hipMemcpy(GPU_in, in_CPU, inlen, hipMemcpyHostToDevice));
        //���Դ��out�ڴ��ʼ��Ϊ0
        checkCudaErrors(hipMemset(GPU_out, 0, GPU_out_buffer_size));
        //���Դ��ϣ���ʼ��
        checkCudaErrors(hipMemset(GPU_HashTable, 0, LZO_WRKMEM_SIZE * threadNum));
        //��outlenAray��ʼ��
        checkCudaErrors(hipMemset(GPU_outlenArray, 0, sizeof(size_t) * threadNum));

   
    finish_rate = 0.25f;
    //����Դ������ֵ
    r.block_size = blocksize;
    r.remainTid = remainTid;
    r.remainSize = remainSize;
    r.in = GPU_in;
    r.out = GPU_out;
    r.hashtable = (unsigned char*)GPU_HashTable;
    r.outlenArray = GPU_outlenArray;
    r.max_out_len = max_out_len_per_thread;
    r.inlen = inlen;
    //////////////////////////////////
    hipEvent_t time1, time2;
    hipEventCreate(&time1);
    hipEventCreate(&time2);
    hipEventRecord(time1);
    cudaLZ << <threadNum/blockdim, blockdim >> > (r);//����ѹ��

   /////////////////////////////
    hipDeviceSynchronize();//�ȴ�ͬ��
    finish_rate = 0.75f;
    hipEventRecord(time2);
    checkCudaErrors(hipEventSynchronize(time1));
    checkCudaErrors(hipEventSynchronize(time2));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, time1, time2));
  //  cout << "CUDA ��ʱ: " << elapsedTime << "ms" << endl;
    //�������������outlenArray������CPU�ڴ�
    size_t* CPU_outlenArray = new size_t[threadNum];
    memset(CPU_outlenArray, 0, sizeof(size_t) * threadNum);
    checkCudaErrors(hipMemcpy(CPU_outlenArray, GPU_outlenArray, sizeof(size_t) * threadNum, hipMemcpyDeviceToHost));
    //for (int i = 0; i < threadNum; ++i)
    //    cout << "outlen=" << CPU_outlenArray[i] << endl;
    //��������������飬�����Ӧ��cpu out����ڴ棬�����Դ濽������
    
    for (int i = 0; i < threadNum; ++i) {
        outputUint temp;
        temp.out_len = CPU_outlenArray[i];
        temp.out_buffer = new unsigned char[temp.out_len + 1024];
        memset(temp.out_buffer, 0, temp.out_len + 1024);
        //��GPU_out[i*max_out_len_perthread,(i+1)*max_out_len_perthread)һ��max_out_len_perthread��С���ڴ�Ϊ��ǰ�߳̿��õ�����ռ�
        checkCudaErrors(hipMemcpy(temp.out_buffer, GPU_out + (i * max_out_len_per_thread), temp.out_len, hipMemcpyDeviceToHost));
        out_CPU.push_back(temp);
       // debug2.push_back(out_CPU[i].out_len);
    }
    finish_rate = 0.90f;
    // hipDeviceSynchronize();//�ȴ�ͬ��
     //�ͷ��Դ�
    hipFree(GPU_in);
    hipFree(GPU_out);
    hipFree(GPU_HashTable);
    hipFree(GPU_outlenArray);
    return int(elapsedTime);
}

//Ӧ�������ⲿ�Ѹ������������ݶ��뵽����������
// CPU_out[block_size*Nparts];
//Nparts:��������
//blocksize:���ݿ��С�����ڽ�ѹ����ȷ�����ڴ�
//�߽��Ͻ����ֱ�ӷ��䵽���С��������϶����ˡ�
bool CallCudaLZO_Decode(vector<inputUint>& compressed_data, unsigned char* CPU_out, int Npats, size_t blocksize)
{
    return true;

}

bool ivan_comp(unsigned char* a, unsigned char* b, size_t len) {

    for (int i = 0; i < len; ++i)
        if (a[i] != b[i])
            return false;

    return true;
}
//__global__ void cudabwt()//(int * arr, int *seq,int n,int alphabetsize) 
//{
//    int strlength;
//    unsigned char* str = NULL;
//    strlength = 6;
//    str = new unsigned char[strlength];
//    str[0] = 'a'; str[1] = 'b'; str[2] = 'a'; str[3] = 'b'; str[4] = 'c';
//    str[strlength - 1] = '$';
//    unsigned char* out = new unsigned char[strlength];
//    int *S=new int [6];
//    int *SA =new int[6];
//    for (int i = 0; i < 6; ++i) {
//        S[i] = 0; SA[i] = 0;
//    }
//    for (int i = 0; i < strlength - 1; ++i) {
//        S[i] = str[i] + 1; //�ַ���ASCII8bit[0,255]����ӳ�䵽ASCII16bit[1,256] �ַ�����С+1
//    }
//    S[strlength - 1] = 0;//���һ��ASCII16 bit :0����С�ַ�
//    for (int i = 0; i < 6; ++i) {
//        printf("%d ", S[i]);
//    }
//    printf("\n��ʼ����cuda_bwt_encodd\n");
//    //cuda_create_suffix_array(SA, S, strlength, 257);
//    //int* SA = new int[strlength];
//    int index =  cuda_bwtEncode(str, out, strlength, S, SA);
//
//    printf("����cuda_create_sa����\n");
//    for (int i = 0; i < 6; ++i) {
//        printf("%d\n", SA[i]);
//    }
//    printf("%s\n", out);
//    unsigned char decode[6];
//    int *ca = new int[257];
//    for (int i = 0; i < 257; ++i)
//        ca[i] = 0;
//    int ja[6] = { 0 };
//    cuda_bwtDecode(out, decode, ca, ja, strlength, index);
//    printf("%s\n", decode);
//}
//N����֧�ַǶ�����ʣ���Ҫ�������� �ô�Ч�ʲ�����
