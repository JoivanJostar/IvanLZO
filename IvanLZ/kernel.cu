﻿#include<iostream>
#include <string>
#include <vector>
#include <fstream>
#include <queue>
#include <cstdio>
#include <assert.h>
#include "global_var.h"
#include <string.h>
#include "Encoder.h"
#include "Decoder.h"
#include <Windows.h>
#include <atlstr.h> 
#include "FileManage.h"
#include <Shlobj.h>
#include <thread>
#include "Console.h"
#include "cuda_lzo.h"
using namespace std;
enum Fuction
{
    ENCODE=1,DECODE
};
enum Moden {
    SINGLE = 1,
    MULTI= 2,
    CUDA = 3
};
//需要封装到类里面




bool IsDeveloper;
int loop_times;
int total_time;


void LoadConfig() {
    FILE* fp;
    fp=freopen("config.ini", "r", stdin);
    if (fp == NULL) {
        cout << "找不到配置文件 config.ini\n";
        exit(0);
    }
  /*  if (scanf("[block_size]=%d;\n", &block_size)) {
        if (block_size <= 0 || block_size>1024) {
            cout << "配置文件损坏\n";
            exit(0);
        }
        g_config.block_size = block_size;
    }
    else {
        cout << "配置文件损坏\n";
        exit(0);
    }
    if (scanf("[BWT]=%d;\n", &use_bwt)) {
        if (use_bwt!=0&&use_bwt!=1) {
            cout << "配置文件损坏\n";
            exit(0);
        }
        g_config.use_bwt = use_bwt;
    }
    else {
        cout << "配置文件损坏\n";
        exit(0);
    }
    if (scanf("[block_dim]=%d;\n", &g_config.block_dim)) {
        if (g_config.block_dim <= 0 || g_config.block_dim > 1024) {
            cout << "配置文件损坏\n";
            exit(0);
        }
    }
    else {
        cout << "配置文件损坏\n";
        exit(0);
    }
    if (scanf("[cuda_block_size]=%d;\n", &g_config.cuda_block_size)) {
        if (g_config.cuda_block_size <= 0 || g_config.cuda_block_size > 1024) {
            cout << "配置文件损坏\n";
            exit(0);
        }
    }
    else {
        cout << "配置文件损坏\n";
        exit(0);
    }*/
    if (scanf("[block_size]=%d;\n[BWT]=%d;\n[block_dim]=%d;\n[cuda_block_size]=%d;\n[threads]=%d;\n[loops]=%d\n[tree]=%d\n", 
        &g_config.block_size,
        &g_config.use_bwt,
        &g_config.block_dim,
        &g_config.cuda_block_size,
        &g_config.thread_num,
        &g_config.loop_time,
        &g_config.print_tree)) {

    }
    else {
        cout << "配置文件损坏\n";
        exit(0);
    }
    fclose(fp);
    
    freopen("CON", "r", stdin);
}
void LoadSysInfo() {
    FILE* fp;
    fp = freopen("sysinfo.xd", "r", stdin);
    if (fp == NULL) {
        cout << "找不到系统文件 sysinfo.xd\n";
        exit(0);
    }
    char line[100] = { 0 };
    gets_s(line);
    int ret=scanf("\n");
    char * pos=strchr(line, '=');
    if (pos == NULL) {
        cout << "系统文件信息异常\n";
        exit(0);
    }
    string strname(pos+1);
    g_system_info.CPU_Name = strname;
    if(!(scanf("[CoreNums]=%d\n", &g_system_info.Core_Num) > 0) ){
        cout << "系统文件信息异常\n";
        exit(0);
    }
    if (!(scanf("[ThreadNums]=%d\n", &g_system_info.Thread_Num) > 0)) {
        cout << "系统文件信息异常\n";
        exit(0);
    }
    if (!(scanf("[DRAM]=%f\n", &g_system_info.DRAM) > 0)) {
        cout << "系统文件信息异常\n";
        exit(0);
    }
    gets_s(line);
    if (!(scanf("%d", &g_system_info.L2CacheSize) > 0)) {
        cout << "获取L2Cache信息失败\n";
        g_system_info.L2CacheSize = 0; 

    }
    gets_s(line);
    gets_s(line);
    if (!(scanf("%d", &g_system_info.L3CacheSize) > 0)) {
        cout << "获取L3Cache信息失败\n";
        g_system_info.L3CacheSize = 0;
    }
    fclose(fp);

    freopen("CON", "r", stdin);
}
int main(int argc,char ** argv) {
   // Test_CUDA_NOBWT();
    //return 0;
    ifstream infile;
    try_get_cpu_info:
    infile.open(".\\sysinfo.xd");
    if (!infile) {
        system("start .\\CPUINFO.exe");
        cout << "正在导入系统信息.........\n";
        Sleep(1000);
        goto try_get_cpu_info;

    }
    infile.close();

    IsDeveloper = true;
    FileManage Mannger;
    ResetGlobalVar();
    loop_times = IsDeveloper ? 10 : 1;
    total_time = 0;

    LoadConfig();
    loop_times = g_config.loop_time;
    LoadSysInfo();
select_En_De:
    cout << "选择功能\n\n【1】压缩\n\n【2】解压\n\n\n";
    string input;
    cin >> input;
    int En_De;
    try {
        En_De = stoul(input);
        if (En_De != 1 && En_De != 2)
            goto select_En_De;
    }
    catch (...) {
        goto select_En_De;
    }
    if (En_De == ENCODE) {
        Console::ClearScreen();
        int input_num;
        vector<string> files;
        string folder;
        cout << "===================================== 压  缩 =======================================" << endl << endl;
    select_2:
        cout << "【1】压缩文件\n\n【2】压缩文件夹\n\n";
        cin >> input;
        try {
            input_num = stoul(input);
            if (input_num != 1 && input_num != 2)
                goto select_2;
        }
        catch (...) {
            goto select_2;
        }
        if (input_num == 1) {
            files = Mannger.SelectFiles();
            if (files.empty())
                goto select_2;
        }
        else {
            folder = Mannger.SelectFolder();
            if (folder == "")
                goto select_2;
        }
        select_6:
        cout << "请选择保存路径\n";
        Sleep(500);
        string SavePath = Mannger.SaveAs();
        if (SavePath == "") {
            cout << "尚未选择目录  可按任意键重新选择输出\n\n";
            system("pause");
            Console::ClearScreen();
            goto select_6;
        }
        auto it1 = SavePath.end() - 1;
        while (*it1 != '\\')
            it1--;
        auto it2 = SavePath.end() - 1;
        while (*it2 != '.')
            it2--;
        string PackageName(it1 + 1, it2);
        string PackageRootPath(SavePath.begin(), it1);

        cout << "\n包名称:" << PackageName << endl;
        if (input_num == 1) {
            ConstructSimpleFileTree(PackageName, files);
        }
        else {
            ConstructFileTree(folder);
        }
        //cout<<"是否要查看文件树的结"
        if (g_config.print_tree) {
            cout << "\n待压缩文件树结构:\n";
            PrintfFileTree(tree);
            cout << "\n\n请确认该文件夹的结构\n\n";
            system("pause");
        }

        Console::ClearScreen();
        //system("cls");
    select_3:
        cout << "请选择压缩模式\n";
        cout << "\n【1】单线程 \n\n【2】CPU多核多线程\n\n【3】GPU CUDA多线程\n\n";
        cin >> input;
        try {
            input_num = stoul(input);
            if (!(input_num <= 3 && input_num >= 1))
                goto select_3;
        }
        catch (...) {
            goto select_3;
        }

        switch (input_num)
        {
        case SINGLE: {
            Console::PrintSystemInfo(g_system_info);
            for (int i = 0; i < loop_times; ++i) {                      
                cout << endl << "正在压缩.......\n";
                Sleep(500);
                g_M = 0;
                finish_rate = 0.0f;
                g_out_totallen = 0;
                g_src_total_len = 0;
                g_time1 = 0;
                g_time2 = 0;
                SET_SINGLE(g_M);
                if (g_config.use_bwt) {
                    SET_BWT(g_M);
                }
                thread t(Encode, SavePath, 0);
                Console::PrintProcedure();
                t.join();
                total_time += (g_time2 - g_time1);
                cout << "\n当前压缩耗时: " << (g_time2 - g_time1) << "ms\n";
                system("pause");
            }              
            break;
        }
        case MULTI: {
            //SET_BWT(g_M);
            Console::PrintSystemInfo(g_system_info);
            cout << endl << "正在压缩.......\n";
            Sleep(500);
            int num = std::thread::hardware_concurrency();
            num = (num > 0) ? (num / 2) : 2;//取核心并发能力一半 胖线程
            num = g_config.thread_num > 0 ? g_config.thread_num : num;
            for (int i = 0; i < loop_times; ++i) {
                g_M = 0;
                finish_rate = 0.0f;
                g_out_totallen = 0;
                g_src_total_len = 0;
                g_time1 = 0;
                g_time2 = 0;
                if (g_config.use_bwt) {
                    SET_BWT(g_M);
                    SET_P_BWT(g_M);
                }
                BootMutilThread_Encode(SavePath, num, g_config.block_size*1024);
                total_time += (g_time2 - g_time1);
                cout << "\n当前压缩耗时: " << (g_time2 - g_time1) << "ms\n";
            }

            break;
        }
        case CUDA: {
            InitGPU();
            for (int i = 0; i < loop_times; ++i) {
                g_M = 0;
                finish_rate = 0.0f;
                g_out_totallen = 0;
                g_src_total_len = 0;
                g_time1 = 0;
                g_time2 = 0;
                SET_CUDA(g_M);
                thread t(Encode, SavePath, g_config.cuda_block_size*1024);
                Console::PrintProcedure();
                t.join();
                total_time += (g_time2 - g_time1);
                cout << "\n当前压缩耗时: " << (g_time2 - g_time1) << "ms\n";
            }

            break;
        }
        default:
            break;
        }

        cout << "平均耗时:" << total_time * 1.0f / loop_times << "ms\n";
        cout << "压缩前:" << g_src_total_len << "字节\n压缩后:" << g_out_totallen << "字节\n";
        if (g_src_total_len == 0)//防止除0
            cout << "压缩率:" << 1.0 << endl;
        else
            cout << "压缩率:" << g_out_totallen * 1.0f / g_src_total_len << endl;
  
        Sleep(1000);
        string cmd = "start " + PackageRootPath;
        system(cmd.c_str());
        cout << "压缩完毕\n";


    }
    else {
        Console::ClearScreen();
        cout << "===================================== 解压缩 ========================================" << endl << endl;
        select_4:
        cout << "\n请选择一个ivan压缩包\n";
        Sleep(500);
        string pack = Mannger.SelectApackage();
        string outPath;
        if (pack != "") {
            select_5:
            cout << "\n请选择输出目录\n";
            Sleep(500);
            outPath = Mannger.SelectFolder();
            if (outPath != "") {
                cout << "正在解压.......\n";
                Sleep(500);
                g_M = 0;
                total_time = 0;
                for (int i = 0; i < loop_times; ++i) {
                    finish_rate = 0;
                    g_time1 = GetTickCount64();
                    BootMutilThread_Decode(pack, outPath, g_config.thread_num);
                    g_time2 = GetTickCount64();
                    cout << "\n当前解压缩耗时: " << (g_time2 - g_time1) << "ms\n";
                    total_time += g_time2 - g_time1;
                }
                cout << "平均耗时:" << total_time * 1.0f / loop_times << "ms\n";
            }
            else {
          
                cout << "尚未选取目录，按任意键后可重新选择\n\n";
                system("pause");
                goto select_5;
            }
               
        }
        else {
            cout << "尚未选取压缩文件，按任意键后可重新选择\n\n";
            system("pause");
            goto select_4;
        }
            
        //cout<<"请选择解压缩模式:"

        
        cout << "\n解压完毕\n";
        Sleep(500);
        string cmd = "start " + outPath;
        system(cmd.c_str());
    }
    cout << "\n\n";
    system("pause");
}

